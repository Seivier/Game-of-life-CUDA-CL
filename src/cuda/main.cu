
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <vector>
#include <random>
#include <fstream>

const int ITER = 500;


using cell = unsigned char;

using namespace std;
#define BLOCK_SIZE 512
#define N 5120
#define M 5120

__global__ void iteration(const cell* in, cell* out, int n, int m) {
    int size = n * m;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        int x1 = i % n;
        int y1 = i - x1;
        int x0 = (x1 + n - 1) % n;
        int x2 = (x1 + 1) % n;
        int y0 = (y1 + n) % (size);
        int y2 = (y1 + size - n) % (size);
        int alive_neighbours = in[x0 + y0] + in[x0 + y1] + in[x0 + y2] + in[x1 + y0] + in[x1 + y2] + in[x2 + y0] + in[x2 + y1] + in[x2 + y2];
        out[x1 + y1] = (alive_neighbours == 3 || (alive_neighbours == 2 && in[x1 + y1])) ? 1 : 0;
    }
}

__global__ void iteration_if(const cell* in, cell* out, int n, int m) {
	int size = n * m;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
		int x1 = i % n;
		int y1 = i - x1;
		int x0 = (x1 + n - 1) % n;
		int x2 = (x1 + 1) % n;
		int y0 = (y1 + n) % (size);
		int y2 = (y1 + size - n) % (size);
		int alive_neighbours = 0;
		if (in[x0 + y0]) alive_neighbours++;
		if (in[x0 + y1]) alive_neighbours++;
		if (in[x0 + y2]) alive_neighbours++;
		if (in[x1 + y0]) alive_neighbours++;
		if (in[x1 + y2]) alive_neighbours++;
		if (in[x2 + y0]) alive_neighbours++;
		if (in[x2 + y1]) alive_neighbours++;
		if (in[x2 + y2]) alive_neighbours++;
		out[x1 + y1] = (alive_neighbours == 3 || (alive_neighbours == 2 && in[x1 + y1])) ? 1 : 0;
	}
}

void random_data(vector<cell>& data) {

    // Init seed
	seed_seq seq{ 123456 };
	random_device rd;
  	default_random_engine eng(seq);
	uniform_int_distribution<int> distr(0, 1);

    for (cell& c : data)
        c = distr(eng);
}

void display(vector<cell> data, int m, int n) {
    int lives = 0;
    for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) 
			lives += data[i * n + j] ? 1 : 0;
	}
    cout << "Live cells: " << lives << endl;
    cout << "Dead cells: " << m * n - lives << endl;
}

int main() {
	for (int tries = 0; tries < 1; tries++)
	{
		cout << "Iteration: " << tries + 1 << '\r';
		const long long size = sizeof(cell) * N * M;
		vector<cell> h_in(N * M), h_out(N * M);
		cell* d_in, * d_out;

		hipMalloc((void**)&d_in, size);
		hipMalloc((void**)&d_out, size);

		// Inicializar la data
		auto t_start = chrono::high_resolution_clock::now();
		random_data(h_in);
		auto t_end = chrono::high_resolution_clock::now();
		auto t_data = chrono::duration<double, milli>(t_end - t_start).count();

		cout << "INITIAL STATE:" << endl;
		display(h_in, N, M);
		cout << endl;

		// Copiar host a device
		t_start = chrono::high_resolution_clock::now();
		hipMemcpy(d_in, h_in.data(), size, hipMemcpyHostToDevice);
		hipMemcpy(d_out, h_out.data(), size, hipMemcpyHostToDevice);
		t_end = chrono::high_resolution_clock::now();
		auto t_host_to_device = chrono::duration<double, milli>(t_end - t_start).count();


		// Correr la simulacion
		t_start = chrono::high_resolution_clock::now();
		for (int i = 0; i < ITER; i++)
		{
			iteration_if <<< (N * M) / BLOCK_SIZE, BLOCK_SIZE >>>(d_in, d_out, N, M);
			swap(d_in, d_out);
		}
		t_end = chrono::high_resolution_clock::now();
		auto t_exec = chrono::duration<double, milli>(t_end - t_start).count();


		// Copiar device a host
		t_start = chrono::high_resolution_clock::now();
		hipMemcpy(h_in.data(), d_in, size, hipMemcpyDeviceToHost);
		hipMemcpy(h_out.data(), d_out, size, hipMemcpyDeviceToHost);
		t_end = chrono::high_resolution_clock::now();
		auto t_device_to_host = chrono::duration<double, milli>(t_end - t_start).count();

		cout << "FINAL STATE: \n";
		display(h_in, N, M);
		cout << endl;

		hipFree(d_in);
		hipFree(d_out);

		cout << "Time to create the data: " << t_data << " ms\n";
		cout << "Time to copy data from host to device: " << t_host_to_device << " ms\n";
		cout << "Time to execute the simulation: " << t_exec << " ms\n";
		cout << "Time to copy data from device to host: " << t_device_to_host << " ms" << endl;
		cout << "Total time: " << t_data + t_exec + t_device_to_host + t_host_to_device << " ms" << endl;
//		fstream file;
//		file.open("cuda_norm.csv", ios::out | ios::app);
//		file << N << "," << M << "," << BLOCK_SIZE << "," << t_data << "," << t_host_to_device << "," << t_exec << ","
//			 << t_device_to_host << "," << t_data + t_exec + t_device_to_host + t_host_to_device << endl;
//		file.close();
	}
	cout << "Complete!" << endl;
	cout << "Data saved in cuda_norm.csv" << endl;
	return 0;
}