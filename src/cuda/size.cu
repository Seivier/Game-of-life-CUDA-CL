#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int device = 0;  // Device ID
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    std::cout << "Maximum Block Size: " << prop.maxThreadsPerBlock << std::endl;

    return 0;
}